#include "hip/hip_runtime.h"
﻿//-----------------------------------------------------------------------------
// File: ImageKernel.cu
// http://people.csail.mit.edu/sparis/bf_course/slides08/03_definition_bf.pdf
// http://www.swageroo.com/wordpress/how-to-program-a-gaussian-blur-without-using-3rd-party-libraries/
// http://rastergrid.com/blog/2010/09/efficient-gaussian-blur-with-linear-sampling/
//
// Desc: 
//
// The Gaussian blur is a type of image-blurring filter that uses a Gaussian function (which also expresses the normal distribution in statistics) 
// for calculating the transformation to apply to each pixel in the image. In two dimensions, it is the product of two Gaussians, one in each dimension.
// In the formula, x is the distance from the origin in the horizontal axis, y is the distance from the origin in the vertical axis, 
// and σ is the standard deviation of the Gaussian distribution. When applied in two dimensions, this formula produces a surface 
// whose contours are concentric circles with a Gaussian distribution from the center point. Values from this distribution are used 
// to build a convolution matrix which is applied to the original image. Each pixel's new value is set to a weighted average of that 
// pixel's neighborhood. The original pixel's value receives the heaviest weight (having the highest Gaussian value) and neighboring pixels 
// receive smaller weights as their distance to the original pixel increases. This results in a blur that preserves boundaries and edges 
// better than other, more uniform blurring filters; see also scale space implementation.	 
// Ref: http://en.wikipedia.org/wiki/Gaussian_blur
//
//-----------------------------------------------------------------------------

#include "stdafx.h"
#include "Utilities.h"
#include "BlurFilter.h"


// Convolution kernel. Must be called once for each channel: red, green, blue.
__global__ 
void gaussian_blur(
	unsigned char* const		blurredChannel,						// return value: blurred channel (either red, green, or blue)
	const unsigned char* const	inputChannel,						// red, green, or blue channel from the original image
	int							rows, 
	int							cols,
	const float* const			filterWeight,						// gaussian filter weights. The weights look like a bell shape.
	int							filterWidth							// number of pixels in x and y directions for calculating average blurring
	)
{
	int r			=  blockIdx.y * blockDim.y + threadIdx.y;		// current row
	int c			=  blockIdx.x * blockDim.x + threadIdx.x;		// current column


	if ((r >= rows) || (c >= cols))
	{
		return;
	}

	int			  half   = filterWidth / 2;
	float		  blur   = 0.f;								// will contained blurred value
	int			  width  = cols - 1;
	int			  height = rows - 1;

	for (int i = -half; i <= half; ++i)					// rows
	{
		for (int j = -half; j <= half; ++j)				// columns
		{
			// Clamp filter to the image border
			int		h		= min(max(r + i, 0), height);
			int		w		= min(max(c + j, 0), width);

			// Blur is a product of current pixel value and weight of that pixel.
			// Remember that sum of all weights equals to 1, so we are averaging sum of all pixels by their weight.
			int		idx		= w + cols * h;											// current pixel index
			float	pixel	= static_cast<float>(inputChannel[idx]);

					idx		= (i + half) * filterWidth + j + half;
			float	weight	= filterWeight[idx];

			blur += pixel * weight;
		}
	}

	blurredChannel[c + r * cols] = static_cast<unsigned char>(blur);
}

// Recombines red, green, and blue channels into an RGB image.
// Alpha channel is set to 255 or opaque.
__global__
void gaussian_recombine_channels(
	uchar4* const				rgba,
	const unsigned char* const	red,
	const unsigned char* const	green,
	const unsigned char* const	blue,
	int							rows,
	int							cols
	)
{
	int x			=  blockIdx.y * blockDim.y + threadIdx.y;		// current row
	int y			=  blockIdx.x * blockDim.x + threadIdx.x;		// current column

	if ((x >= rows) || (y >= cols))
	{
		return;
	}

	int idx			= y + cols * x;		// current pixel index

	// Copy channels to the local variables
	unsigned char r = red[idx];
	unsigned char g = green[idx];
	unsigned char b = blue[idx];

	// Combine, setting alpha to 255
	uchar4 pixel = make_uchar4(r, g, b, 255);

	// Update image
	rgba[idx] = pixel;
}

// Kernel separates rgba image into red, green, blue channels
__global__ 
void gaussian_separate_channels(
	unsigned char* const red, 
	unsigned char* const green, 
	unsigned char* const blue, 
	const uchar4* const  rgba, 
	int					 rows, 
	int					 cols
	)
{
	int r			=  blockIdx.y * blockDim.y + threadIdx.y;		// current row
	int c			=  blockIdx.x * blockDim.x + threadIdx.x;		// current column

	if ((r >= rows) || (c >= cols))
	{
		return;
	}

	int idx			= c + cols * r;		// current pixel index

	red  [idx]		= rgba[idx].x;
	green[idx]		= rgba[idx].y;
	blue [idx]		= rgba[idx].z;
}

// Applies gaussian blur to an r8g8b8a8 image.
// Returns blurredimage.
void RunGaussianBlurKernel(
	uchar4* const			blurredImage,					// Return value: blurred rgba image with alpha set to 255 or opaque.
	const uchar4* const		originalImage,
	unsigned char* const	red,							// red channel from the original image
	unsigned char* const	green,							// green channel from the original image
	unsigned char* const	blue,							// blue channel from the original image
	unsigned char* const	redBlurred,						// red channel from the blurred image
	unsigned char* const	greenBlurred,					// green channel from the blurred image
	unsigned char* const	blueBlurred,					// blue channel from the blurred image
	const float* const		filterWeight,					// gaussian filter weights. The weights look like a bell shape.
	int						filterWidth,					// number of pixels in x and y directions for calculating average blurring
	int						rows,							// image size: number of rows
	int						cols							// image size: number of columns
	)
{
	const char* func = "RunGaussianBlurKernel";

	hipError_t hr = hipSuccess;

	static const int BLOCK_WIDTH = 32;						// threads per block; because we are setting 2-dimensional block, the total number of threads is 32^2, or 1024
															// 1024 is the maximum number of threads per block for modern GPUs.

	int x = static_cast<int>(ceilf(static_cast<float>(cols) / BLOCK_WIDTH));
	int y = static_cast<int>(ceilf(static_cast<float>(rows) / BLOCK_WIDTH));

	const dim3 grid (x, y, 1);								// number of blocks
	const dim3 block(BLOCK_WIDTH, BLOCK_WIDTH, 1);			// block width: number of threads per block

	// Separate RGBA image into different color channels
	gaussian_separate_channels<<<grid, block>>>(
		red,
		green,
		blue,
		originalImage, 
		rows, 
		cols
		);
		
	hr = hipDeviceSynchronize();																CHECK_CUDA_ERROR(hr, func, "separate_channels kernel failed.");

	// Call convolution kernel for the total of 3 times, once for each channel
	gaussian_blur<<< grid, block >>>(
		redBlurred,
		red,
		rows,
		cols,
		filterWeight,
		filterWidth
		);

	hr = hipDeviceSynchronize();																CHECK_CUDA_ERROR(hr, func, "gaussian_blur kernel failed - red channel");

	gaussian_blur<<<grid, block>>>(
		greenBlurred,
		green,
		rows,
		cols,
		filterWeight,
		filterWidth
		);

	hr = hipDeviceSynchronize();																CHECK_CUDA_ERROR(hr, func, "gaussian_blur kernel failed - green channel");

	gaussian_blur<<<grid, block>>>(
		blueBlurred,
		blue,
		rows,
		cols,
		filterWeight,
		filterWidth
		);

	hr = hipDeviceSynchronize();																CHECK_CUDA_ERROR(hr, func, "gaussian_blur kernel failed - blue channel");

	// Recombine red, green,and blue channels into an RGB image
	gaussian_recombine_channels<<<grid, block>>>(
		blurredImage,
		redBlurred,
		greenBlurred,
		blueBlurred,
		rows,
		cols
		);

	hr = hipDeviceSynchronize();																CHECK_CUDA_ERROR(hr, func, "gaussian_recombine_channels kernel failed.");
}
